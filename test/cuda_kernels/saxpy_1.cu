
#include <hip/hip_runtime.h>

extern "C" __global__ 
void saxpy_1(float a, float *x, float *out, size_t n) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    out[tid] = a * x[tid];
  }
}
