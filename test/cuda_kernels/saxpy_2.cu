
#include <hip/hip_runtime.h>

extern "C" __global__ 
void saxpy_2(float *x, float *y, float *out, size_t n) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    out[tid] = x[tid] + y[tid];
  }
}
